/* Includes, system */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* Includes, cuda */
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>

int main(int argc, char **argv) {
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);
}
